#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "./polybenchUtilFuncts.h"
//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

/* Problem size. */
#define NX 32768
#define NY 32768

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 1024
#define DIM_THREAD_BLOCK_Y 1

/* spin */
#define DIM_ROW (NX/DIM_THREAD_BLOCK_X)       //32768/1024=32
#define DIM_COLUMN (NY/DIM_THREAD_BLOCK_Y)    //32768
#define DIM_BLOCK DIM_ROW*DIM_COLUMN		  
#define DIM_BLOCK_VECTOR DIM_BLOCK/4		

#define NUM_SM 80
#define NUM_SM_HtoD 4
#define OFFSET NUM_SM_HtoD * DIM_THREAD_BLOCK_X
#define NUM_SM_COMPUTE_tmp 38
#define NUM_SM_COMPUTE_y   38

#define IN_CHUNK_SIZE 32
#define IN_CHUNK NX/IN_CHUNK_SIZE 			//1024
#define IN_CHUNK_OFFSET OFFSET*IN_CHUNK_SIZE*2  	    


#ifndef M_PI
#define M_PI 3.14159
#endif

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

__device__ int flag_global_read(volatile int * flag, int rid)
{
	return(flag[rid]);
}

void init_array(DATA_TYPE *x, DATA_TYPE *A)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*(j)) / NX;
		}
	}
}


void compareResults(DATA_TYPE *z, DATA_TYPE *z_outputFromGpu)
{
	int i, fail;
	fail = 0;

	for (i=0; i<NY; i++)
	{
		if (percentDiff(z[i], z_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}		
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void atax_kernel(DATA_TYPE *A, DATA_TYPE *A_host,  DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *tmp, int *flag_A, int* flag_tmp)
{
     
	if (blockIdx.x < NUM_SM_HtoD ){  //copy kernel HtoD
		const int idx = threadIdx.x;
                const int bidx = blockIdx.x;
		int chunk_offset;
		for (int i = 0; i<IN_CHUNK;i++ ){
			chunk_offset=i*IN_CHUNK_OFFSET;

			for (int k = (chunk_offset+bidx*1024+idx);k < ( chunk_offset+IN_CHUNK_OFFSET ) ; k+= OFFSET ){
			reinterpret_cast<double2*>(A)[k] = reinterpret_cast<double2*>(A_host)[k];
			}

			__syncthreads();
			

			__threadfence();

			if ( idx < IN_CHUNK_SIZE ){
            		atomicAdd(&flag_A[IN_CHUNK_SIZE*i+idx],1);
     		   	}
		}
	} else if (blockIdx.x < (NUM_SM_HtoD + NUM_SM_COMPUTE_tmp)){
	// compute tmp

		int i = (blockIdx.x - NUM_SM_HtoD) * blockDim.x + threadIdx.x;
        	if (i < NX)
        	{
			while(  flag_global_read(flag_A, i) != 4 );

                	for(int j=0; j < NY; j++)
               		{
                        	tmp[i] += A[i * NY + j] * x[j];
               		}
			

			__threadfence();
			atomicOr(&flag_tmp[i],1);
        	}
	

	} else {//if (blockIdx.x < (NUM_SM_HtoD + NUM_SM_COMPUTE_tmp + NUM_SM_COMPUTE_y)){
	//compute y
	
	int j = (blockIdx.x - NUM_SM_HtoD - NUM_SM_COMPUTE_tmp) * blockDim.x + threadIdx.x;

        	if (j < NY)
       		 {
                	for(int i=0; i < NX; i++)
                	{
			
			while(  flag_global_read(flag_tmp,i) == 0 );

                       	 y[j] += A[i * NY + j] * tmp[i];
                	}	
        	}
	}
}



void atax_cpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
	int i,j;
	
	for (i= 0; i < NY; i++)
	{
    	y[i] = 0;
	}
  
	for (i = 0; i < NX; i++)
 	{
      	tmp[i] = 0;

      	for (j = 0; j < NY; j++)
		{
			tmp[i] = tmp[i] + A[i*NY + j] * x[j];
		}
		
      	for (j = 0; j < NY; j++)
		{
			y[j] = y[j] + A[i*NY + j] * tmp[i];
		}
    }
}


void ataxGpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp, DATA_TYPE* y_outputFromGpu)
{

	DATA_TYPE *A_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NX);
	
	int *flag_A, *flag_tmp;
	hipMalloc((void **)&flag_A, sizeof(int) * NX);
	hipMalloc((void **)&flag_tmp, sizeof(int) * NX);
	
	hipMemset(flag_A, 0, sizeof(int) * NX);
	hipMemset(flag_tmp, 0, sizeof(int) * NX);
	
	//hipMemset(y_gpu, 0, sizeof(DATA_TYPE) * NY);
	//hipMemset(tmp_gpu, 0, sizeof(DATA_TYPE) * NX);


	hipEvent_t start,stop;
        float elapsedTimeInMs = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(NUM_SM), (size_t)1 );

	void *kernelArgs[] = {
                (void *)&A_gpu,             (void *)&A,
                (void *)&x_gpu,             (void *)&y_gpu,
                (void *)&tmp_gpu,           (void *)&flag_A,
		(void *)&flag_tmp
        };	


	hipEventRecord(start);

        hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
        hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
        hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);

	hipLaunchCooperativeKernel((void*)atax_kernel, grid, block, kernelArgs, 0, NULL);
	//atax_kernel<<< grid, block >>>(A_gpu,A,x_gpu,y_gpu, tmp_gpu,flag_A, flag_tmp);
	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);
	

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTimeInMs, start, stop);
	fprintf(stdout,"GPU RunTime= %.2f Ms \n",  elapsedTimeInMs);


	// debug copy array A

	/*	
	DATA_TYPE* A_debug;
	hipHostAlloc((void **)&A_debug, sizeof(DATA_TYPE) * NX * NY, hipHostMallocPortable);
	hipMemcpy(A_debug, A_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost);	
	compareResults(A,A_debug);
	*/


	//debug tmp array
	/*
	DATA_TYPE* tmp_debug;
	hipHostAlloc((void **)&tmp_debug, sizeof(DATA_TYPE) * NX, hipHostMallocPortable);
	hipMemcpy(tmp_debug, tmp_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);
	atax_cpu(A, x, y, tmp);
	compareResults(tmp, tmp_debug);
	*/
	
	//debug flag_tmp;
	/*
	int* flag_tmp_debug;
        hipHostAlloc((void **)&flag_tmp_debug, sizeof(int) * NX, hipHostMallocPortable);
	hipMemcpy(flag_tmp_debug, flag_tmp, sizeof(int) * NX, hipMemcpyDeviceToHost);
	for (int i = 0 ; i < NX; i++){
	fprintf(stdout, "%d", flag_tmp_debug[i]);
	}
	*/

	hipFree(A_gpu);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(tmp_gpu);
}


int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* x;
	DATA_TYPE* y;
	DATA_TYPE* y_outputFromGpu;
	DATA_TYPE* tmp;

	/*
	A = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
	x = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y_outputFromGpu = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));
	*/

	hipHostAlloc((void **)&A, sizeof(DATA_TYPE) * NX * NY, hipHostMallocPortable);
	hipHostAlloc((void **)&x, sizeof(DATA_TYPE) * NY, hipHostMallocPortable);
	hipHostAlloc((void **)&y, sizeof(DATA_TYPE) * NY, hipHostMallocPortable);
	hipHostAlloc((void **)&y_outputFromGpu, sizeof(DATA_TYPE) * NY, hipHostMallocPortable);
	hipHostAlloc((void **)&tmp, sizeof(DATA_TYPE) * NX, hipHostMallocPortable);
	

	

	init_array(x, A);

	GPU_argv_init();
	ataxGpu(A, x, y, tmp, y_outputFromGpu);
	
		
	t_start = rtclock();
	atax_cpu(A, x, y, tmp);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(y, y_outputFromGpu);
	
	hipFree(A);
	hipFree(x);
	hipFree(y);
	hipFree(y_outputFromGpu);
	hipFree(tmp);

  	return 0;
}

